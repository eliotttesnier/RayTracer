#include "hip/hip_runtime.h"
#include "CUDAKernel.cuh"
#include <hip/hip_runtime.h>
#include <>
#include <stdio.h>
#include <stdexcept>
#include "../Math/Point3D.hpp"
#include "../Math/Vector3D.hpp"
#include "../Math/Ray.hpp"
#include "../Graphic/Color.hpp"
#include "../Primitives/Sphere/Sphere.hpp"
#include "../Primitives/Triangles/Triangles.hpp"
#include "../Lights/ILight.hpp"
#include "../Graphic/Camera.hpp"

namespace RayTracer {
namespace CUDA {

// Variables pour stocker les données sur le GPU
Sphere* d_spheres = nullptr;
Triangle* d_triangles = nullptr;
Light* d_lights = nullptr;
int numSpheres = 0;
int numTriangles = 0;
int numLights = 0;
RenderParams renderParams;

// Fonctions utilitaires pour le CUDA
__device__ float dot(const Vector3D& a, const Vector3D& b) {
    return a.x * b.x + a.y * b.y + a.z * b.z;
}

__device__ float length(const Vector3D& v) {
    return sqrtf(dot(v, v));
}

__device__ Vector3D normalize(const Vector3D& v) {
    float len = length(v);
    if (len > 0.0f) {
        return {v.x / len, v.y / len, v.z / len};
    }
    return v;
}

__device__ Vector3D cross(const Vector3D& a, const Vector3D& b) {
    return {
        a.y * b.z - a.z * b.y,
        a.z * b.x - a.x * b.z,
        a.x * b.y - a.y * b.x
    };
}

__device__ Vector3D subtract(const Point3D& a, const Point3D& b) {
    return {a.x - b.x, a.y - b.y, a.z - b.z};
}

__device__ Point3D add(const Point3D& p, const Vector3D& v) {
    return {p.x + v.x, p.y + v.y, p.z + v.z};
}

__device__ Ray generateRay(const Camera& camera, float u, float v) {
    // Calcul de la direction du rayon en fonction des coordonnées de l'écran
    float aspectRatio = camera.aspectRatio;
    float scale = tanf(camera.fov * 0.5f * 3.14159f / 180.0f);
    
    float imageX = (2.0f * u - 1.0f) * aspectRatio * scale;
    float imageY = (1.0f - 2.0f * v) * scale;
    
    Vector3D direction = {
        camera.right.x * imageX + camera.up.x * imageY + camera.direction.x,
        camera.right.y * imageX + camera.up.y * imageY + camera.direction.y,
        camera.right.z * imageX + camera.up.z * imageY + camera.direction.z
    };
    
    return {camera.position, normalize(direction)};
}

__device__ bool intersectSphere(const Ray& ray, const Sphere& sphere, HitData& hitData) {
    Vector3D oc = {
        ray.origin.x - sphere.center.x,
        ray.origin.y - sphere.center.y,
        ray.origin.z - sphere.center.z
    };
    
    float a = dot(ray.direction, ray.direction);
    float b = 2.0f * dot(oc, ray.direction);
    float c = dot(oc, oc) - sphere.radius * sphere.radius;
    float discriminant = b*b - 4*a*c;
    
    if (discriminant < 0) {
        return false;
    }
    
    float t = (-b - sqrtf(discriminant)) / (2.0f * a);
    if (t < 0.001f) {
        t = (-b + sqrtf(discriminant)) / (2.0f * a);
        if (t < 0.001f) {
            return false;
        }
    }
    
    hitData.distance = t;
    hitData.point = {
        ray.origin.x + ray.direction.x * t,
        ray.origin.y + ray.direction.y * t,
        ray.origin.z + ray.direction.z * t
    };
    
    Vector3D normal = {
        (hitData.point.x - sphere.center.x) / sphere.radius,
        (hitData.point.y - sphere.center.y) / sphere.radius,
        (hitData.point.z - sphere.center.z) / sphere.radius
    };
    
    hitData.normal = normalize(normal);
    hitData.color = sphere.color;
    hitData.hit = true;
    
    return true;
}

__device__ bool intersectTriangle(const Ray& ray, const Triangle& triangle, HitData& hitData) {
    // Implémentation de l'algorithme de Möller-Trumbore
    Vector3D edge1 = subtract(triangle.p2, triangle.p1);
    Vector3D edge2 = subtract(triangle.p3, triangle.p1);
    Vector3D h = cross(ray.direction, edge2);
    float a = dot(edge1, h);
    
    if (a > -0.0001f && a < 0.0001f) {
        return false; // Le rayon est parallèle au triangle
    }
    
    float f = 1.0f / a;
    Vector3D s = subtract(ray.origin, triangle.p1);
    float u = f * dot(s, h);
    
    if (u < 0.0f || u > 1.0f) {
        return false;
    }
    
    Vector3D q = cross(s, edge1);
    float v = f * dot(ray.direction, q);
    
    if (v < 0.0f || u + v > 1.0f) {
        return false;
    }
    
    float t = f * dot(edge2, q);
    
    if (t > 0.001f) {
        hitData.distance = t;
        hitData.point = {
            ray.origin.x + ray.direction.x * t,
            ray.origin.y + ray.direction.y * t,
            ray.origin.z + ray.direction.z * t
        };
        
        hitData.normal = triangle.normal;
        hitData.color = triangle.color;
        hitData.hit = true;
        return true;
    }
    
    return false;
}

__device__ HitData traceRay(
    const Ray& ray,
    Sphere* spheres, int numSpheres,
    Triangle* triangles, int numTriangles,
    int depth
) {
    HitData closestHit;
    closestHit.hit = false;
    closestHit.distance = FLT_MAX;
    
    // Vérifier l'intersection avec toutes les sphères
    for (int i = 0; i < numSpheres; i++) {
        HitData hitData;
        hitData.hit = false;
        
        if (intersectSphere(ray, spheres[i], hitData)) {
            if (hitData.distance < closestHit.distance) {
                closestHit = hitData;
            }
        }
    }
    
    // Vérifier l'intersection avec tous les triangles
    for (int i = 0; i < numTriangles; i++) {
        HitData hitData;
        hitData.hit = false;
        
        if (intersectTriangle(ray, triangles[i], hitData)) {
            if (hitData.distance < closestHit.distance) {
                closestHit = hitData;
            }
        }
    }
    
    return closestHit;
}

__device__ bool isInShadow(
    const Point3D& hitPoint,
    const Vector3D& lightDir,
    float lightDistance,
    Sphere* spheres, int numSpheres,
    Triangle* triangles, int numTriangles
) {
    // Bias pour éviter les ombres acnéiques (shadow acne)
    const float SHADOW_BIAS = 0.001f;
    Point3D shadowOrigin = {
        hitPoint.x + lightDir.x * SHADOW_BIAS,
        hitPoint.y + lightDir.y * SHADOW_BIAS,
        hitPoint.z + lightDir.z * SHADOW_BIAS
    };
    
    Ray shadowRay = {shadowOrigin, lightDir};
    
    // Vérifier les intersections pour déterminer si le point est dans l'ombre
    for (int i = 0; i < numSpheres; i++) {
        HitData hitData;
        if (intersectSphere(shadowRay, spheres[i], hitData)) {
            if (hitData.distance > 0.0f && (lightDistance < 0.0f || hitData.distance < lightDistance)) {
                return true;
            }
        }
    }
    
    for (int i = 0; i < numTriangles; i++) {
        HitData hitData;
        if (intersectTriangle(shadowRay, triangles[i], hitData)) {
            if (hitData.distance > 0.0f && (lightDistance < 0.0f || hitData.distance < lightDistance)) {
                return true;
            }
        }
    }
    
    return false;
}

__device__ Color calculateColor(
    const HitData& hitData,
    const Ray& ray,
    Light* lights, int numLights,
    Sphere* spheres, int numSpheres,
    Triangle* triangles, int numTriangles,
    int depth, int maxDepth
) {
    if (!hitData.hit || depth >= maxDepth) {
        return {0.0f, 0.0f, 0.0f, 1.0f};
    }
    
    Color finalColor = {0.0f, 0.0f, 0.0f, 1.0f};
    
    // Facteurs de phong
    const float ka = 0.1f;  // Ambient
    const float kd = 0.7f;  // Diffuse
    const float ks = 0.2f;  // Specular
    const float shininess = 32.0f;
    
    // Calculer la contribution de chaque lumière
    for (int i = 0; i < numLights; i++) {
        Light light = lights[i];
        
        // Lumière ambiante (toujours présente)
        if (light.type == 0) { // Ambient
            finalColor.r += light.color.r * light.intensity * ka;
            finalColor.g += light.color.g * light.intensity * ka;
            finalColor.b += light.color.b * light.intensity * ka;
            continue;
        }
        
        // Calculer la direction de la lumière
        Vector3D lightDir;
        float lightDistance = -1.0f;
        
        if (light.type == 1) { // Directional
            lightDir = {-light.direction.x, -light.direction.y, -light.direction.z};
        } else { // Point
            lightDir = {
                light.position.x - hitData.point.x,
                light.position.y - hitData.point.y,
                light.position.z - hitData.point.z
            };
            lightDistance = length(lightDir);
            lightDir = {
                lightDir.x / lightDistance,
                lightDir.y / lightDistance,
                lightDir.z / lightDistance
            };
        }
        
        // Vérifier si le point est dans l'ombre
        if (!isInShadow(hitData.point, lightDir, lightDistance, spheres, numSpheres, triangles, numTriangles)) {
            // Éclairage diffus
            float diffuse = max(0.0f, dot(hitData.normal, lightDir));
            
            // Éclairage spéculaire
            Vector3D reflectDir = {
                2.0f * dot(hitData.normal, lightDir) * hitData.normal.x - lightDir.x,
                2.0f * dot(hitData.normal, lightDir) * hitData.normal.y - lightDir.y,
                2.0f * dot(hitData.normal, lightDir) * hitData.normal.z - lightDir.z
            };
            Vector3D viewDir = {-ray.direction.x, -ray.direction.y, -ray.direction.z};
            float specular = powf(max(0.0f, dot(reflectDir, viewDir)), shininess);
            
            // Atténuation pour les lumières ponctuelles
            float attenuation = 1.0f;
            if (light.type == 2) { // Point
                attenuation = 1.0f / (1.0f + 0.09f * lightDistance + 0.032f * lightDistance * lightDistance);
            }
            
            // Ajouter la contribution de la lumière
            finalColor.r += (diffuse * kd + specular * ks) * light.color.r * light.intensity * attenuation;
            finalColor.g += (diffuse * kd + specular * ks) * light.color.g * light.intensity * attenuation;
            finalColor.b += (diffuse * kd + specular * ks) * light.color.b * light.intensity * attenuation;
        }
    }
    
    // Appliquer la couleur de l'objet
    finalColor.r *= hitData.color.r;
    finalColor.g *= hitData.color.g;
    finalColor.b *= hitData.color.b;
    
    // Correction gamma simple
    finalColor.r = min(1.0f, finalColor.r);
    finalColor.g = min(1.0f, finalColor.g);
    finalColor.b = min(1.0f, finalColor.b);
    
    // Conversion en espace RGB 0-255
    finalColor.r = powf(finalColor.r, 1.0f/2.2f) * 255.0f;
    finalColor.g = powf(finalColor.g, 1.0f/2.2f) * 255.0f;
    finalColor.b = powf(finalColor.b, 1.0f/2.2f) * 255.0f;
    
    return finalColor;
}

__global__ void renderKernel(
    Color* outputBuffer,
    int width, int height,
    Sphere* spheres, int numSpheres,
    Triangle* triangles, int numTriangles,
    Light* lights, int numLights,
    Camera camera,
    RenderParams params
) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (x >= width || y >= height) {
        return;
    }
    
    int pixelIndex = y * width + x;
    
    // Calcul des coordonnées UV normalisées
    float u = (float)x / (float)(width - 1);
    float v = (float)y / (float)(height - 1);
    
    // Génération du rayon
    Ray ray = generateRay(camera, u, v);
    
    // Anti-aliasing par échantillonnage (supersampling)
    Color pixelColor = {0.0f, 0.0f, 0.0f, 1.0f};
    
    if (params.useAntialiasing) {
        for (int sy = 0; sy < params.samples; sy++) {
            for (int sx = 0; sx < params.samples; sx++) {
                float su = ((float)x + (sx + 0.5f) / params.samples) / (float)(width - 1);
                float sv = ((float)y + (sy + 0.5f) / params.samples) / (float)(height - 1);
                
                Ray sampleRay = generateRay(camera, su, sv);
                HitData hitData = traceRay(sampleRay, spheres, numSpheres, triangles, numTriangles, 0);
                
                Color sampleColor = calculateColor(
                    hitData, sampleRay, 
                    lights, numLights, 
                    spheres, numSpheres, 
                    triangles, numTriangles, 
                    0, params.maxDepth
                );
                
                pixelColor.r += sampleColor.r;
                pixelColor.g += sampleColor.g;
                pixelColor.b += sampleColor.b;
            }
        }
        
        float invSamples = 1.0f / (params.samples * params.samples);
        pixelColor.r *= invSamples;
        pixelColor.g *= invSamples;
        pixelColor.b *= invSamples;
    } else {
        // Sans anti-aliasing
        HitData hitData = traceRay(ray, spheres, numSpheres, triangles, numTriangles, 0);
        pixelColor = calculateColor(
            hitData, ray, 
            lights, numLights, 
            spheres, numSpheres, 
            triangles, numTriangles, 
            0, params.maxDepth
        );
    }
    
    // Écriture dans le buffer de sortie
    outputBuffer[pixelIndex] = pixelColor;
}

// Implémentation des fonctions d'interface avec le CPU
void convertToCUDAStructures(
    const std::vector<std::shared_ptr<IPrimitive>>& primitives,
    const std::vector<std::shared_ptr<ILight>>& lights,
    const std::shared_ptr<RayTracer::Camera>& camera,
    const int width, const int height,
    std::vector<Sphere>& spheres,
    std::vector<Triangle>& triangles,
    std::vector<Light>& cudaLights,
    Camera& cudaCamera
) {
    // Conversion des sphères
    for (const auto& primitive : primitives) {
        auto sphere = std::dynamic_pointer_cast<RayTracer::primitive::Sphere>(primitive);
        if (sphere) {
            Sphere cudaSphere;
            
            // Conversion de la position
            Math::Point3D pos = sphere->getPosition();
            cudaSphere.center = {static_cast<float>(pos._x), static_cast<float>(pos._y), static_cast<float>(pos._z)};
            
            // Conversion du rayon
            cudaSphere.radius = static_cast<float>(sphere->getRadius());
            
            // Conversion de la couleur
            Graphic::color_t color = sphere->getColor();
            cudaSphere.color = {static_cast<float>(color.r), static_cast<float>(color.g), static_cast<float>(color.b), static_cast<float>(color.a)};
            
            // Propriétés matérielles (simplifiées)
            cudaSphere.reflection = 0.0f;  // À compléter avec les vraies propriétés
            cudaSphere.refraction = 0.0f;
            cudaSphere.transparency = 0.0f;
            
            spheres.push_back(cudaSphere);
        }
        
        auto triangle = std::dynamic_pointer_cast<RayTracer::primitive::Triangles>(primitive);
        if (triangle) {
            Triangle cudaTriangle;
            
            // À compléter pour les triangles
            triangles.push_back(cudaTriangle);
        }
    }
    
    // Conversion des lumières
    for (const auto& light : lights) {
        Light cudaLight;
        
        // Position
        Math::Point3D pos = light->getPosition();
        cudaLight.position = {static_cast<float>(pos._x), static_cast<float>(pos._y), static_cast<float>(pos._z)};
        
        // Direction (pour les lumières directionnelles)
        Math::Vector3D dir = light->getDirection();
        cudaLight.direction = {static_cast<float>(dir._x), static_cast<float>(dir._y), static_cast<float>(dir._z)};
        
        // Couleur
        auto [r, g, b] = light->getColor();
        cudaLight.color = {static_cast<float>(r) / 255.0f, static_cast<float>(g) / 255.0f, static_cast<float>(b) / 255.0f, 1.0f};
        
        // Intensité
        float intensity;
        light->getIntensity(intensity);
        cudaLight.intensity = intensity;
        
        // Type
        std::string type = light->getType();
        if (type == "AmbientLight") {
            cudaLight.type = 0;
        } else if (type == "DirectionalLight") {
            cudaLight.type = 1;
        } else if (type == "PositionLight" || type == "PointLight") {
            cudaLight.type = 2;  // Point light
        } else {
            cudaLight.type = 2;  // Point light par défaut
        }
        
        cudaLights.push_back(cudaLight);
    }
    
    // Conversion de la caméra
    Math::Point3D camPos = camera->getPosition();
    cudaCamera.position = {static_cast<float>(camPos._x), static_cast<float>(camPos._y), static_cast<float>(camPos._z)};
    
    cudaCamera.fov = static_cast<float>(camera->getFov());
    cudaCamera.aspectRatio = static_cast<float>(width) / static_cast<float>(height);
    
    // Construction des vecteurs de base de la caméra
    Math::Vector3D camDirection = camera->getForward();
    Math::Vector3D camUp = camera->getUp();
    Math::Vector3D camRight = camera->getRight();
    
    cudaCamera.direction = {static_cast<float>(camDirection._x), static_cast<float>(camDirection._y), static_cast<float>(camDirection._z)};
    cudaCamera.up = {static_cast<float>(camUp._x), static_cast<float>(camUp._y), static_cast<float>(camUp._z)};
    cudaCamera.right = {static_cast<float>(camRight._x), static_cast<float>(camRight._y), static_cast<float>(camRight._z)};
}

void initGPUData(
    const std::vector<Sphere>& spheres,
    const std::vector<Triangle>& triangles,
    const std::vector<Light>& lights,
    const Camera& camera,
    const RenderParams& params
) {
    // Libérer d'abord la mémoire si nécessaire
    freeGPUData();
    
    // Stocker le nombre d'objets
    numSpheres = spheres.size();
    numTriangles = triangles.size();
    numLights = lights.size();
    renderParams = params;
    
    // Allouer et copier les données sur le GPU
    if (numSpheres > 0) {
        hipMalloc(&d_spheres, numSpheres * sizeof(Sphere));
        hipMemcpy(d_spheres, spheres.data(), numSpheres * sizeof(Sphere), hipMemcpyHostToDevice);
    }
    
    if (numTriangles > 0) {
        hipMalloc(&d_triangles, numTriangles * sizeof(Triangle));
        hipMemcpy(d_triangles, triangles.data(), numTriangles * sizeof(Triangle), hipMemcpyHostToDevice);
    }
    
    if (numLights > 0) {
        hipMalloc(&d_lights, numLights * sizeof(Light));
        hipMemcpy(d_lights, lights.data(), numLights * sizeof(Light), hipMemcpyHostToDevice);
    }
}

void freeGPUData() {
    if (d_spheres) {
        hipFree(d_spheres);
        d_spheres = nullptr;
    }
    
    if (d_triangles) {
        hipFree(d_triangles);
        d_triangles = nullptr;
    }
    
    if (d_lights) {
        hipFree(d_lights);
        d_lights = nullptr;
    }
}

void renderCUDA(Color* outputBuffer, int width, int height) {
    // Allouer le buffer de sortie sur le GPU
    Color* d_outputBuffer;
    hipMalloc(&d_outputBuffer, width * height * sizeof(Color));
    
    // Configurer les dimensions des blocs et de la grille
    dim3 blockSize(16, 16);
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x, (height + blockSize.y - 1) / blockSize.y);
    
    // Lancer le kernel
    renderKernel<<<gridSize, blockSize>>>(
        d_outputBuffer,
        width, height,
        d_spheres, numSpheres,
        d_triangles, numTriangles,
        d_lights, numLights,
        Camera(),  // Cette caméra doit être initialisée correctement
        renderParams
    );
    
    // Attendre que tous les threads aient terminé
    hipDeviceSynchronize();
    
    // Copier le résultat du GPU vers le CPU
    hipMemcpy(outputBuffer, d_outputBuffer, width * height * sizeof(Color), hipMemcpyDeviceToHost);
    
    // Libérer la mémoire
    hipFree(d_outputBuffer);
}

} // namespace CUDA
} // namespace RayTracer
